#include <iostream>
#include "../src/BruteForceGpu.cuh"
#include "../src/KdTreeGpu.cuh"
#include "../src/BallTreeGpu.cuh"
#include "Tests.cuh"


int main(int argc, char const *argv[]) {
    
    hipFree(0);
    constexpr uint dims = 3;
    constexpr uint pointCount = 1000000;
    constexpr uint qryCount = 1000000;
    constexpr uint k = 10;


    // QryCount increase speed tests
    KnnTrees::Array<float, dims>* points = KnnTrees::pinnedMalloc<KnnTrees::Array<float, dims>>(pointCount);
    generateRandomPoints<dims>(points, pointCount, 1.0);
    KnnTrees::Array<float, dims>* dPoints = gpuCopy(points, pointCount);
    constexpr uint QryCounts[] = {250000, 500000, 1000000, 2500000, 5000000, 10000000};
    forLoop<6>([&](auto i) {
        constexpr uint qryCount = QryCounts[i];
        KnnTrees::Array<float, dims>* queryPoints = KnnTrees::pinnedMalloc<KnnTrees::Array<float, dims>>(qryCount);
        generateRandomPoints<dims>(queryPoints, qryCount, 1.0);
        KnnTrees::Array<float, dims>* dQueryPoints = gpuCopy(queryPoints, qryCount);
        KnnTrees::Query<dims, k> query = {queryPoints, qryCount, 5.9f, nullptr, nullptr};
        query.rIndexes = KnnTrees::pinnedMalloc<KnnTrees::Array<int, k>>(qryCount);
        query.rDistances = KnnTrees::pinnedMalloc<KnnTrees::Array<float, k>>(qryCount);
        KnnTrees::Query<dims, k> dQuery = {dQueryPoints, qryCount, 5.9f, nullptr, nullptr};
        KnnTrees::Cuda::check(hipMalloc(&dQuery.rIndexes, sizeof(KnnTrees::Array<int, k>) * qryCount));
        KnnTrees::Cuda::check(hipMalloc(&dQuery.rDistances, sizeof(KnnTrees::Array<float, k>) * qryCount));
        float kdTime = 0.0f; float ballTime = 0.0f;
        for (uint j = 0; j < 6; j++) {
            // KdTree GPU
            KnnTrees::KdTreeGpu<dims, dims, 32> kdg(dPoints, pointCount, 3);
            startTime();
            kdg.batchKnn<k, 32, false, false, true>(dQuery);
            if (j != 0) kdTime += getTime();
            // BallTree GPU
            KnnTrees::BallTreeGpu<dims, 32> ballg(dPoints, pointCount, 1);            
            startTime();
            ballg.batchKnn<k, 16, true, true, true>(dQuery);
            if (j != 0) ballTime += getTime();
        }
        std::cout << "KdTreeGpu - qries: " << qryCount << ": " << (kdTime / 5) << "ms" << std::endl;
        std::cout << "BallTreeGpu - qries: " << qryCount << ": " << (ballTime / 5) << "ms" << std::endl;
        KnnTrees::Cuda::check(hipFree(dQuery.rDistances));
        KnnTrees::Cuda::check(hipFree(dQuery.rIndexes));
        KnnTrees::pinnedFree(query.rDistances);
        KnnTrees::pinnedFree(query.rIndexes);
        KnnTrees::pinnedFree(queryPoints);
        KnnTrees::Cuda::check(hipFree(dQueryPoints));
    });
    KnnTrees::pinnedFree(points);
    KnnTrees::Cuda::check(hipFree(dPoints));


    // PointCount increase speed tests
    KnnTrees::Array<float, dims>* queryPoints = KnnTrees::pinnedMalloc<KnnTrees::Array<float, dims>>(qryCount);
    generateRandomPoints<dims>(queryPoints, qryCount, 1.0);
    KnnTrees::Array<float, dims>* dQueryPoints = gpuCopy(queryPoints, qryCount);
    constexpr uint PointCounts[] = {250000, 500000, 1000000, 2500000, 5000000, 10000000};
    forLoop<6>([&](auto i) {
        constexpr uint pointCount = PointCounts[i];
        KnnTrees::Array<float, dims>* points = KnnTrees::pinnedMalloc<KnnTrees::Array<float, dims>>(pointCount);
        generateRandomPoints<dims>(points, pointCount, 1.0);
        KnnTrees::Array<float, dims>* dPoints = gpuCopy(points, pointCount);
        KnnTrees::Query<dims, k> query = {queryPoints, qryCount, 5.9f, nullptr, nullptr};
        query.rIndexes = KnnTrees::pinnedMalloc<KnnTrees::Array<int, k>>(qryCount);
        query.rDistances = KnnTrees::pinnedMalloc<KnnTrees::Array<float, k>>(qryCount);
        KnnTrees::Query<dims, k> dQuery = {dQueryPoints, qryCount, 5.9f, nullptr, nullptr};
        KnnTrees::Cuda::check(hipMalloc(&dQuery.rIndexes, sizeof(KnnTrees::Array<int, k>) * qryCount));
        KnnTrees::Cuda::check(hipMalloc(&dQuery.rDistances, sizeof(KnnTrees::Array<float, k>) * qryCount));
        float kdTime = 0.0f; float ballTime = 0.0f;
        for (uint j = 0; j < 6; j++) {
            // KdTree GPU
            KnnTrees::KdTreeGpu<dims, dims, 32> kdg(dPoints, pointCount, 3);
            startTime();
            kdg.batchKnn<k, 32, false, false, true>(dQuery);
            if (j != 0) kdTime += getTime();
            // BallTree GPU
            KnnTrees::BallTreeGpu<dims, 32> ballg(dPoints, pointCount, 1);
            startTime();
            ballg.batchKnn<k, 16, true, true, true>(dQuery);
            if (j != 0) ballTime += getTime();
        }
        std::cout << "KdTreeGpu - points: " << pointCount << ": " << (kdTime / 5) << "ms" << std::endl;
        std::cout << "BallTreeGpu - points: " << pointCount << ": " << (ballTime / 5) << "ms" << std::endl;
        KnnTrees::Cuda::check(hipFree(dQuery.rDistances));
        KnnTrees::Cuda::check(hipFree(dQuery.rIndexes));
        KnnTrees::pinnedFree(query.rDistances);
        KnnTrees::pinnedFree(query.rIndexes);
        KnnTrees::pinnedFree(points);
        KnnTrees::Cuda::check(hipFree(dPoints));
    });
    KnnTrees::pinnedFree(queryPoints);
    KnnTrees::Cuda::check(hipFree(dQueryPoints));


    return 0;
}
